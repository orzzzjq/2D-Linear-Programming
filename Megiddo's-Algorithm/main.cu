#include "hip/hip_runtime.h"
#include "helper.cuh"
#include "io.cuh"

using namespace std;

answer compute(inputs input);

int main() {
	// get the input data
	inputs input = read_from_file("./data");
	// get the answer
	long long t1 = clock();
	answer ans = compute(input);
	double compute_time = (double)(clock() - t1) / CLOCKS_PER_SEC;
	// 3display result and free memory
	char * ans_string = generate_ans_string(ans);
	printf("%s", ans_string);
	printf("Compute time: %fs\n", compute_time);

	getchar();
	return 0;
}

__host__ __device__
bool ccw(point a, point b) {
	return strictly_larger(a.pos_x * b.pos_y - a.pos_y * b.pos_x, 0);
}

struct rotation
{
	double a, b;
	__host__ __device__
	rotation(double a, double b)
		: a(a), b(b) {}

	__host__ __device__
	line operator()(line l)
	{
		line res;
		point obj_x(b, -a);
		point obj_y(a, b);
		point p1, p2, p_y(0, 1);
		if (l.slope_value < 1 && l.slope_value > -1) {
			line x1(1, 0, 10000);
			line x2(1, 0,-10000);
			p1 = generate_intersection_point(l, x1);
			p2 = generate_intersection_point(l, x2);

		}
		else {
			line y1(0, 1, 10000);
			line y2(0, 1,-10000);
			p1 = generate_intersection_point(l, y1);
			p2 = generate_intersection_point(l, y2);
		}

		point p3(p1.shadow(obj_x), p1.shadow(obj_y));
		point p4(p2.shadow(obj_x), p2.shadow(obj_y));

		res = generate_line_from_2points(p3, p4);
		if (strictly_larger(l.param_c, 0) ^ strictly_larger(res.param_c, 0)) {
			res.param_a *= -1;
			res.param_b *= -1;
			res.param_c *= -1;
		}
		res.upper = (res.param_b < 0);
		res.save = 1;
		return res;
	}
};

struct is_upper {
	__host__ __device__
	bool operator() (line l) { return l.upper; }
};

struct is_lower {
	__host__ __device__
	bool operator() (line l) { return !l.upper; }
};

struct tmpPoint
{
	point p;
	int lineNo1, lineNo2;
	__host__ __device__
	tmpPoint() {}
};

struct intersection
{
	thrust::device_vector<int>::iterator lineNo;
	thrust::device_vector<tmpPoint>::iterator tmp_points;
	thrust::device_vector<line>::iterator rotated_lines;
	
	__host__ __device__
	intersection(thrust::device_vector<int>::iterator lineNo,
		thrust::device_vector<tmpPoint>::iterator tmp_points,
		thrust::device_vector<line>::iterator rotated_lines)
		: lineNo(lineNo), tmp_points(tmp_points), rotated_lines(rotated_lines) {}
	
	__host__ __device__
	bool operator() (int idx)
	{
		if (idx % 2) return 0;
		int lineNo1 = *(lineNo + idx), lineNo2 = *(lineNo + idx + 1);
		tmpPoint new_tmpPoint;
		new_tmpPoint.p = generate_intersection_point(*(rotated_lines+lineNo1), *(rotated_lines + lineNo2));
		new_tmpPoint.lineNo1 = lineNo1;
		new_tmpPoint.lineNo2 = lineNo2;
		*(tmp_points + idx / 2) = new_tmpPoint;
		return 0;
	}
};

struct comparator {
	__host__ __device__
	bool operator() (tmpPoint a, tmpPoint b) {
		return a.p.pos_x < b.p.pos_x;
	}
};

struct line_cmp
{
	double x;
	__host__ __device__
	line_cmp(double x) : x(x) {}

	__host__ __device__
	bool operator() (line line1, line line2)
	{
		double y1 = (-line1.param_a * x + line1.param_c) / line1.param_b;
		double y2 = (-line2.param_a * x + line2.param_c) / line2.param_b;
		return y1 < y2;
	}
};

struct judge
{
	bool left;
	double median_x;
	thrust::device_vector<line>::iterator rotated_lines;

	__host__ __device__
	judge(bool left, double median_x,
		thrust::device_vector<line>::iterator rotated_lines)
		: left(left), median_x(median_x), rotated_lines(rotated_lines) {}

	__host__ __device__
	bool operator() (tmpPoint tmp_p)
	{
		point p = tmp_p.p;
		line line1 = *(rotated_lines + tmp_p.lineNo1);
		line line2 = *(rotated_lines + tmp_p.lineNo2);
		if (line1.upper ^ line2.upper); // one is I+ and the other is I-
		else if (left) { // on the left side
			if (p.pos_x - EPS < median_x);
			else {
				if (line1.upper) { // both are I-, remove the line with smaller slope
					if (strictly_less(line1.slope_value, line2.slope_value)) line1.save = 0;
					if (strictly_less(line2.slope_value, line1.slope_value)) line2.save = 0;
				}
				else { // both are I+, remove the line with larger slope
					if (strictly_larger(line1.slope_value, line2.slope_value)) line1.save = 0;
					if (strictly_larger(line2.slope_value, line1.slope_value)) line2.save = 0;
				}
			}
		}
		else { // on the right side
			if (p.pos_x + EPS > median_x);
			else {
				if (line1.upper) { // both are I-, remove the line with larger slope
					if (strictly_larger(line1.slope_value, line2.slope_value)) line1.save = 0;
					if (strictly_larger(line2.slope_value, line1.slope_value)) line2.save = 0;
				}
				else { // both are I+, remove the line with smaller slope
					if (strictly_less(line1.slope_value, line2.slope_value)) line1.save = 0;
					if (strictly_less(line2.slope_value, line1.slope_value)) line2.save = 0;
				}
			}
		}
		*(rotated_lines + tmp_p.lineNo1) = line1;
		*(rotated_lines + tmp_p.lineNo2) = line2;
		return 0;
	}
};

struct remove_or_not
{
	thrust::device_vector<line>::iterator rotated_lines;

	__host__ __device__
	remove_or_not(thrust::device_vector<line>::iterator rotated_lines)
		: rotated_lines(rotated_lines) {}

	__host__ __device__
	bool operator() (int idx)
	{
		line l = *(rotated_lines + idx);
		return l.save;
	}
};

struct saved
{
	__host__ __device__
	bool operator() (line l)
	{
		return l.save;
	}
};

#define lines input.lines

answer compute(inputs input) {
	answer ans;
	int num = input.number;

	// copy data to gpu
	thrust::device_vector <line> d_lines = lines;
	thrust::device_vector <line> rotated_lines(num);
	thrust::device_vector <line> upper_lines(num);
	thrust::device_vector <line> lower_lines(num);

	// rotation
	thrust::transform(d_lines.begin(), d_lines.end(), rotated_lines.begin(),
		rotation(input.obj_function_param_a, input.obj_function_param_b));
	
	// divide the lines into two parts
	thrust::partition(rotated_lines.begin(), rotated_lines.end(), is_upper());
	thrust::host_vector <line> h_lines(num);
	thrust::copy(rotated_lines.begin(), rotated_lines.begin() + num, h_lines.begin());
	thrust::copy_if(rotated_lines.begin(), rotated_lines.end(), upper_lines.begin(), is_upper());
	thrust::copy_if(rotated_lines.begin(), rotated_lines.end(), lower_lines.begin(), is_lower());
	int upper_num = thrust::count_if(rotated_lines.begin(), rotated_lines.end(), is_upper());
	int lower_num = num - upper_num;

	if (!lower_num) { //no answer
		ans.answer_b = MAXFLOAT;
		ans.intersection_point = point(0, 0);
		ans.line1 = generate_line_from_abc(0, 0, 0);
		ans.line2 = generate_line_from_abc(0, 0, 0);
		return ans;
	}
	
	thrust::device_vector <bool> useless(num);
	thrust::device_vector <int> lineNo(num);
	thrust::sequence(lineNo.begin(), lineNo.end());
	thrust::device_vector <int> Idx(num);
	thrust::sequence(Idx.begin(), Idx.end());
	thrust::device_vector <tmpPoint> tmp_points(num);
	// remove n/4 lines each time
	// until there are less than 10 lines
	while (num > 10) {
		// Partition the lines into pairs
		// and compute the intersection point of every pair
		int point_num = num / 2;
		thrust::transform(Idx.begin(), Idx.begin() + num, useless.begin(),
			intersection(lineNo.begin(), tmp_points.begin(), rotated_lines.begin()));

		// sort the points in the x-order
		// find the median point
		thrust::sort(tmp_points.begin(), tmp_points.begin() + point_num, comparator());
		tmpPoint median_p = tmp_points[point_num / 2];
		double median_x = median_p.p.pos_x;

		// find the highest I+ and the lowest I-
		int lower_idx = thrust::max_element(lower_lines.begin(),
			lower_lines.begin() + lower_num, line_cmp(median_x)) - lower_lines.begin();
		int upper_idx = thrust::min_element(upper_lines.begin(), 
			upper_lines.begin() + upper_num, line_cmp(median_x)) - upper_lines.begin();

		// determine which side of the test line gives the answer
		pair <double, double> H, U, L;
		line max_lower = lower_lines[lower_idx];
		line min_upper = upper_lines[upper_idx];
		L.first = (- max_lower.param_a * median_x + max_lower.param_c) / max_lower.param_b;
		L.second = max_lower.slope_value;
		if (upper_num == 0) U.first = MAXFLOAT, U.second = 0;
		else {
			U.first = (- min_upper.param_a * median_x + min_upper.param_c) / min_upper.param_b;
			U.second = min_upper.slope_value;
		}
		H.first = U.first - L.first;
		H.second = U.second - L.second;
		
		bool left, noAns = 0;
		if (equals(H.first, 0)) {                   // h(x)=0
			if (L.second < 0) left = 0;             // right
			else left = 1;                          // left   
		}
		else if (H.first < 0) {                     // h(x)<0
			if (equals(H.second, 0)) noAns = 1;     // no answer
			else if (H.second < 0) left = 1;        // left
			else left = 0;                          // right
		}
		else {                                      // h(x)>0
			if (L.second > 0) left = 1;             // left
			else left = 0;                          // right
		}

		if (noAns) { // no answer
			ans.answer_b = MAXFLOAT;
			ans.intersection_point = point(0, 0);
			ans.line1 = generate_line_from_abc(0, 0, 0);
			ans.line2 = generate_line_from_abc(0, 0, 0);
			return ans;
		}

		// judge every point and mark some useless lines
		thrust::transform(tmp_points.begin(), tmp_points.begin() + point_num, 
			useless.begin(), judge(left, median_x, rotated_lines.begin()));

		// remove the useless lines
		thrust::copy_if(Idx.begin(), Idx.end(), lineNo.begin(), remove_or_not(rotated_lines.begin()));
		num = thrust::count_if(rotated_lines.begin(), rotated_lines.end(), saved());
	}
	
	// copy the line number to cpu
	thrust::host_vector <int> h_lineNo(num);
	thrust::copy(lineNo.begin(), lineNo.begin() + num, h_lineNo.begin());

	// compute the answer
	double res_x, min_y = MAXFLOAT;
	int lineNo1 = 0, lineNo2 = 0;
	for (int i = 0; i < num; i++) {
		for (int j = i + 1; j < num; j++) {
			line l1 = h_lines[h_lineNo[i]];
			line l2 = h_lines[h_lineNo[j]];
			if (is_parallel(l1, l2)) continue;
			point p = generate_intersection_point(l1, l2);
			bool flag = 1;
			for (int k = 0; k < num; k++) {
				if (i == k || j == k) continue;
				line l3 = h_lines[h_lineNo[k]];
				double _y = (-l3.param_a * p.pos_x + l3.param_c) / l3.param_b;
				if (l3.upper && strictly_larger(p.pos_y, _y)) { flag = 0; break; }
				if ((!l3.upper) && strictly_less(p.pos_y, _y)) { flag = 0; break; }
			}
			if (flag) {
				if (p.pos_y < min_y) {
					res_x = p.pos_x;
					min_y = p.pos_y;
					lineNo1 = h_lineNo[i];
					lineNo2 = h_lineNo[j];
				}
			}
		}
	}

	printf("Rotated point: (%f, %f)\n", res_x, min_y);

	// rotate back
	line l1 = h_lines[lineNo1];
	line l2 = h_lines[lineNo2];

	point obj_x(input.obj_function_param_b, input.obj_function_param_a);
	point obj_y(-input.obj_function_param_a, input.obj_function_param_b);
	point p1, p2, p3, p4;
	if (l1.slope_value < 1 && l1.slope_value > -1) {
		line x1(1, 0, 10000);
		line x2(1, 0, -10000);
		p1 = generate_intersection_point(l1, x1);
		p2 = generate_intersection_point(l1, x2);

	}
	else {
		line y1(0, 1, 10000);
		line y2(0, 1, -10000);
		p1 = generate_intersection_point(l1, y1);
		p2 = generate_intersection_point(l1, y2);
	}
	p3.pos_x = p1.shadow(obj_x), p3.pos_y = p1.shadow(obj_y);
	p4.pos_x = p2.shadow(obj_x), p4.pos_y = p2.shadow(obj_y);
	l1 = generate_line_from_2points(p3, p4);
	if (l2.slope_value < 1 && l2.slope_value > -1) {
		line x1(1, 0, 10000);
		line x2(1, 0, -10000);
		p1 = generate_intersection_point(l2, x1);
		p2 = generate_intersection_point(l2, x2);

	}
	else {
		line y1(0, 1, 10000);
		line y2(0, 1, -10000);
		p1 = generate_intersection_point(l2, y1);
		p2 = generate_intersection_point(l2, y2);
	}
	p3.pos_x = p1.shadow(obj_x), p3.pos_y = p1.shadow(obj_y);
	p4.pos_x = p2.shadow(obj_x), p4.pos_y = p2.shadow(obj_y);
	l2 = generate_line_from_2points(p3, p4);
	
	ans.line1 = l1;
	ans.line2 = l2;
	ans.intersection_point = generate_intersection_point(l1, l2);
	ans.answer_b = ans.intersection_point.pos_x * input.obj_function_param_a
		+ ans.intersection_point.pos_y * input.obj_function_param_b;
	return ans;
}
